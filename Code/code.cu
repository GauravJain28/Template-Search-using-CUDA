#include <bits/stdc++.h>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;


__global__ void kernel(int* data_img, int* query_img, float* d_ans, float th1, float th2, int m, int n, int mq, int nq, float imqsum){
    
    //thread id
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    //start point
    int x0 = tid % n;
    int y0 = tid / n;

    //sin45 and cos45
    float sin45 = 0.70710678118;
    float cos45 = 0.70710678118;

    for(int orientation = 0; orientation<3;orientation++){
        d_ans[y0*n*3 + x0*3 + orientation] = FLT_MAX;
    
        float hcomp_x,hcomp_y;
        float vcomp_x,vcomp_y;

        float x1,y1,x2,y2,x3,y3;
        int a0,b0,a1,b1,a2,b2,a3,b3;

        //0-degree
        if (orientation == 0){
            x1 = x0;
            y1 = y0 + mq-1;

            x2 = x0 + nq-1;
            y2 = y1;

            x3 = x2;
            y3 = y0;

            a0 = (int) x0;
            b0 = (int) y0;

            a1 = a0;
            b1 = (int) y1;

            a2 = (int) x2;
            b2 = b1;

            a3 = a2;
            b3 = b0;

            hcomp_x = 1;
            vcomp_x = 0;

            hcomp_y = 0;
            vcomp_y = 1;
        }

        // -45 degree
        else if(orientation == 1){
            x1 = x0 + (mq-1)*sin45;
            y1 = y0 + (mq-1)*cos45;

            x2 = x0 + (nq-1)*cos45 + (mq-1)*sin45;
            y2 = y0 + (mq-1)*cos45 - (nq-1)*sin45;

            x3 = x0 + (nq-1)*cos45;
            y3 = y0 - (nq-1)*sin45;

            a0 = (int) x0;
            b0 = (int) y3;

            a1 = a0;
            b1 = (int) y1;

            a2 = (int) x2;
            b2 = b1;

            a3 = a2;
            b3 = b0;

            hcomp_x = cos45;
            vcomp_x = sin45;

            hcomp_y = -sin45;
            vcomp_y = cos45;

        }

        // 45 degree
        else{
            x1 = x0 - (mq-1)*sin45;
            y1 = y0 + (mq-1)*cos45;

            x2 = x0 + (nq-1)*cos45 - (mq-1)*sin45;
            y2 = y0 + (mq-1)*cos45 + (nq-1)*sin45;

            x3 = x0 + (nq-1)*cos45;
            y3 = y0 + (nq-1)*sin45;

            a0 = (int) x1;
            b0 = (int) y0;

            a1 = a0;
            b1 = (int) y2;

            a2 = (int) x3;
            b2 = b1;

            a3 = a2;
            b3 = b0;

            hcomp_x = cos45;
            vcomp_x = -sin45;

            hcomp_y = sin45;
            vcomp_y = cos45;
        }

        //out of bounds check
        if(b0 < 0 || a0 < 0 || b2 > m-1 || a2 > n-1){
            //return;
        }
        else{

        //filtering
        float imsum = 0;

        for (int i = b0; i <= b1; i++){
            for (int j= a0 ; j <= a3; j++){
                float temp = 0;
                temp += data_img[i*n*3 + j*3];
                temp += data_img[i*n*3 + j*3 + 1];
                temp += data_img[i*n*3 + j*3 + 2];
                //temp = temp/(3*(a3-a0+1)*(b1-b0+1));
                imsum += temp;

            }
        }

        imsum = imsum/(3*(a3-a0+1)*(b1-b0+1));
        imsum = abs(imsum - imqsum);

        // std::cout << "Average: " << imsum << endl;

        if (imsum >= th2){
            //return;
        }
        else{

        //rmsd

        float rmd = 0.0;

        for (int i = 0; i < mq; i++){
            for (int j = 0; j < nq; j++){
                float x =  (x0 + hcomp_x*j + vcomp_x*i);
                float y =  (y0 + hcomp_y*j + vcomp_y*i);

                int bx0 = (int) x;
                int by0 = (int) y;

                int bx1 = bx0;
                int by1 = by0 + 1;

                int bx2 = bx0 + 1;
                int by2 = by1;

                int bx3 = bx2;
                int by3 = by0;

                float gx = x - bx0;
                float gy = y - by0;

                float red = data_img[by0*n*3 + bx0*3]*(1-gx)*(1-gy) 
                          + data_img[by1*n*3 + bx1*3]*(1-gx)*(gy)
                          + data_img[by2*n*3 + bx2*3]*(gx)*(gy)
                          + data_img[by3*n*3 + bx3*3]*(gx)*(1-gy);

                float green = data_img[by0*n*3 + bx0*3+1]*(1-gx)*(1-gy) 
                          + data_img[by1*n*3 + bx1*3+1]*(1-gx)*(gy)
                          + data_img[by2*n*3 + bx2*3+1]*(gx)*(gy)
                          + data_img[by3*n*3 + bx3*3+1]*(gx)*(1-gy);

                float blue = data_img[by0*n*3 + bx0*3+2]*(1-gx)*(1-gy) 
                          + data_img[by1*n*3 + bx1*3+2]*(1-gx)*(gy)
                          + data_img[by2*n*3 + bx2*3+2]*(gx)*(gy)
                          + data_img[by3*n*3 + bx3*3+2]*(gx)*(1-gy);

                float qred = query_img[i*nq*3 + j*3];
                float qgreen = query_img[i*nq*3 + j*3 + 1];
                float qblue = query_img[i*nq*3 + j*3 + 2];

                rmd += (qred-red)*(qred-red) + (qgreen - green)*(qgreen - green) + (qblue-blue)*(qblue-blue);

            }
        }

        rmd = rmd/(3*nq*mq);

        rmd = sqrt(rmd);
        d_ans[y0*n*3 + x0*3 + orientation] = rmd;
            }    
        }
    }
}





int main(int argc, char const *argv[]){

    string data_img_p = argv[1];
    string query_img_p = argv[2];
    int num = stoi(argv[5]);
    float th1 = stof(argv[3]);
    float th2 = stof(argv[4]);

    ifstream data_file; 
    data_file.open(data_img_p,ios::in);

    int data_m;
    int data_n;

    string word;
    data_file >> word;
    data_m = stoi(word);
    data_file >> word;
    data_n = stoi(word);

    int* data_img = new int[data_m*data_n*3];

    for(int i=0;i<data_m;i++){
        for(int j=0;j<data_n*3;j++){
            data_file >> word;
            data_img[(data_m-i-1)*data_n*3 + j] = stoi(word);
        }
    }
    
    ifstream query_file; 
    query_file.open(query_img_p,ios::in);

    int query_m;
    int query_n;

    query_file >> word;
    query_m = stoi(word);
    query_file >> word;
    query_n = stoi(word);

    int* query_img = new int[query_m*query_n*3];

    for(int i=0;i<query_m;i++){
        for(int j=0;j<query_n*3;j++){
            query_file >> word;
            query_img[(query_m-i-1)*query_n*3 + j] = stoi(word);
        }
    }

    float imqsum = 0;

    for (int i = 0; i < query_m*query_n*3; i++){
        imqsum += query_img[i];
    }
    imqsum = imqsum/(query_m*query_n*3);

    // cout<<data_img[(840)*1200*3 + 900*3]<<" "<<query_img[0]<<endl;

    size_t data_size = 3*data_m*data_n*sizeof(int);
    size_t query_size = 3*query_m*query_n*sizeof(int);
    size_t ans_size = 3*data_m*data_n*sizeof(float);

    float* ans = (float*)malloc(ans_size);
    
    //cuda memory allocation
    int* d_data_img;
    int* d_query_img;
    float* d_ans;

    hipMalloc(&d_data_img, data_size);
    hipMalloc(&d_query_img, query_size);
    hipMalloc(&d_ans, ans_size);
    
    hipMemcpy(d_data_img, data_img, data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_query_img, query_img, query_size, hipMemcpyHostToDevice);

    int blkDim = (data_m*data_n)/1024 + 1;
    int thdDim = 1024;

    kernel<<<blkDim,thdDim>>>(d_data_img,d_query_img,d_ans,th1,th2,data_m,data_n,query_m,query_n,imqsum);
    //cudaDeviceSynchronize();
    
    //float* ans = (float*)malloc(data_size);
    hipMemcpy(ans, d_ans, ans_size, hipMemcpyDeviceToHost);
  
    //std::cout << ans[841*1200*3+3*900] << " " << ans[1] << " "<< ans[2] << endl;
    
    
    float minrmsd = FLT_MAX;
    int minrow,mincol,minor;
    
    //std::cout<<"started\n";
    
    for(int i = 0; i < data_m;i++){
        for (int j = 0; j < data_n; j++){
            if(ans[i*data_n*3 + j*3]<minrmsd){
                    minrow = i;
                    mincol = j;
                    minor = 0;
                    minrmsd = ans[i*data_n*3 + j*3];
            }
            if(ans[i*data_n*3 + j*3+1]<minrmsd){
                    minrow = i;
                    mincol = j;
                    minor = 1;
                    minrmsd = ans[i*data_n*3 + j*3+1];
            }
            if(ans[i*data_n*3 + j*3+2]<minrmsd){
                    minrow = i;
                    mincol = j;
                    minor = 2;
                    minrmsd = ans[i*data_n*3 + j*3+2];
            }
        }
    }
    //std::cout<<"ended\n";
    
    
    int angle[3] = {0,-45,45};
    ofstream myfile;
    myfile.open ("output.txt");
    myfile << minrow<<" "<<mincol<<" "<<angle[minor]<<"\n";
    
    myfile.close();
    
            
    
}